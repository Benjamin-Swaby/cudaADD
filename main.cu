
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
#include <stdlib.h>
#include "string.h"


__global__ void add(int *a , int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index]; 

}


void randomints(int *arr, int size)
{
    for(int i = 0; i < size; i++)
    {   
        int rand = random();
        arr[i] = rand;
        
    }
    
}


void printRESULT(int *result, char *name)
{   
    printf("--------------%s-----------------\n",name);
    for(int i = 0; result[i] != NULL; i++)
    {
        printf("%d , ",result[i]);

    }
    printf("\n");


}


#define N (16384*16384)
#define THREADS_PER_BLOCK 512
int main(int argc, char *argv[])
{
    //init time
    srand(time(NULL));
    int show = 0;

    //check for cli args
    if(argc > 1)
    {
        printf("%s",argv[1]);
        if(strcmp(argv[1],"show") == 0)
        {
            show = 1;
        }
    }


    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
    
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    
    a = (int *)malloc(size); randomints(a,N);
    b = (int *)malloc(size); randomints(b,N);
    c = (int *)malloc(size);
    
    if(show)
    {
        printRESULT(a,"a");
        printRESULT(b,"b");
    } 

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(N+THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c,N);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    if(show)
    { 
        printRESULT(c,"c");
    }

    free(a); free(b), free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;



}
